#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

extern "C++"
{
#include "../include/project/hyperrectangle.hpp"
#include "../include/project/asset.hpp"
#include "../include/project/finance_computation.hpp"
#include "../include/project/asset.hpp"
#include "../include/project/finance_montecarlo.hpp"
#include "../include/project/optionparameters.hpp"
#include "../include/project/finance_inputmanager.hpp"
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ void generateGaussianNumbers( float *total_value, float *total_squared_value,
                                        const float *assets_returns, const float *assets_std_devs, long long int n,
                                        float *assets_closing_values, int strike_price, long long int seed, float *predicted_assets_prices )
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // TODO 4 is the maximum number of assets
    float result = 0.0;
    float rnd_daily_return = 0.0;
    float closing_value;

    if (tid < n)
    {
        hiprandState_t state;
        hiprand_init(seed, tid, 0, &state); // Moved outside the loop

        for( size_t asset_idx = 0; asset_idx < 4; asset_idx++)
        {
            closing_value = assets_closing_values[asset_idx];
            
            float return_value = hiprand_normal(&state);
            rnd_daily_return = assets_returns[asset_idx] + assets_std_devs[asset_idx] * return_value;
            closing_value = closing_value * (1 + rnd_daily_return);

            result += closing_value;
            atomicAdd(&predicted_assets_prices[asset_idx], closing_value);
        }

        if (result > strike_price)
            {
                result = result - strike_price;
            }else result = 0.0;

        atomicAdd(&total_value[tid % 100000], result);
        atomicAdd(&total_squared_value[tid % 100000], result * result);
    }
}




__global__ void printFunction(long long int n, char *function, const double *coefficients, int number_of_coefficients)
{
    printf("Function: %s\n", function);
    printf("n: %ld\n", n);
    for (size_t i = 0; i < number_of_coefficients; ++i)
    {
        printf("Coefficient[%ld]: %f\n", i, coefficients[i]);
    }
    printf("Number of coefficients: %d\n", number_of_coefficients);
}



extern std::pair<double, double> kernel_wrapper(long long int n, const std::string &function, HyperRectangle &hyperrectangle,
                                                const std::vector<const Asset *> &assetPtrs, double std_dev_from_mean, double *variance,
                                                std::vector<double> coefficients, double strike_price, long long int seed)
{
    auto start = std::chrono::high_resolution_clock::now();
    dim3 threads_per_block = 256;
    dim3 number_of_blocks = (n + threads_per_block.x - 1) / threads_per_block.x;

    uint num_assets = assetPtrs.size();
    // Create and copy function and coefficients to device
    char *d_function;
    size_t function_size = function.size() + 1; // Include the null terminator
    gpuErrchk( hipMalloc((void **)&d_function, function_size * sizeof(char)) );
    gpuErrchk( hipMemcpy(d_function, function.c_str(), function_size * sizeof(char), hipMemcpyHostToDevice) );

    double *d_coefficients;
    gpuErrchk( hipMalloc((void **)&d_coefficients, coefficients.size() * sizeof(double)) );
    gpuErrchk( hipMemcpy(d_coefficients, coefficients.data(), coefficients.size() * sizeof(double), hipMemcpyHostToDevice) );
    

    // Call the CUDA kernel to print the function and coefficients
    // printf("Calling CUDA kernel!\n");
     printFunction<<<1, 1>>>(n, d_function, d_coefficients, coefficients.size());
    // printf("CUDA kernel finished!\n");

    // Save the assets main data
    float *d_assets_returns;
    float *d_assets_std_devs;
    float *d_assets_last_values;
    gpuErrchk( hipMalloc((void **)&d_assets_returns, num_assets * sizeof(float)) );
    gpuErrchk( hipMalloc((void **)&d_assets_std_devs, num_assets * sizeof(float)) );
    gpuErrchk( hipMalloc((void **)&d_assets_last_values, num_assets * sizeof(float)) );

    for (size_t i = 0; i < num_assets; i++)
    {
        float return_mean = static_cast<float>(assetPtrs[i]->getReturnMean());
        gpuErrchk( hipMemcpy(&d_assets_returns[i], &return_mean, sizeof(float), hipMemcpyHostToDevice) );

        float return_std_dev = static_cast<float>(assetPtrs[i]->getReturnStdDev());
        gpuErrchk( hipMemcpy(&d_assets_std_devs[i], &return_std_dev, sizeof(float), hipMemcpyHostToDevice) );

        float last_value = static_cast<float>(assetPtrs[i]->getLastRealValue());
        gpuErrchk( hipMemcpy(&d_assets_last_values[i], &last_value, sizeof(float), hipMemcpyHostToDevice) );
    }

    
    double total_value = 0.0;
    double total_squared_value = 0.0;

    float *d_total_value, *d_total_squared_value;
    gpuErrchk( hipMalloc(&d_total_value, 100000 * sizeof(float)) );
    gpuErrchk( hipMalloc(&d_total_squared_value, 100000 * sizeof(float)) );

    float predicted_assets_prices[num_assets];;
    float *d_predicted_assets_prices;
    gpuErrchk( hipMalloc(&d_predicted_assets_prices, num_assets * sizeof(float)) );

    generateGaussianNumbers<<<number_of_blocks, threads_per_block>>>( d_total_value, d_total_squared_value, d_assets_returns, d_assets_std_devs, n, d_assets_last_values, strike_price, seed, d_predicted_assets_prices);
    hipDeviceSynchronize();

    float host_total_value[100000];
    float host_total_squared_value[100000];
    gpuErrchk( hipMemcpy(host_total_value, d_total_value, 100000 * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(host_total_squared_value, d_total_squared_value, 100000 * sizeof(float), hipMemcpyDeviceToHost) );

    for (size_t i = 0; i <  100000; i++)
    {
        total_value += static_cast<double>( host_total_value[i] );
        total_squared_value += static_cast<double>( host_total_squared_value[i] );
    }


    float host_assets_prices[num_assets];
    gpuErrchk( hipMemcpy(host_assets_prices, d_predicted_assets_prices, num_assets * sizeof(float), hipMemcpyDeviceToHost) );

    for( size_t i = 0; i < num_assets; ++i )
    {
        predicted_assets_prices[i] = ( host_assets_prices[i]/n );
        std::cout << "The predicted future price (30 days) of one " << assetPtrs[i]->getName() << " stock is " << predicted_assets_prices[i] << std::endl;
    }


    double option_payoff = total_value / n;

    // calculate the variance
    *variance = total_squared_value/n - (total_value / n) * (total_value/ n);
    *variance = sqrt(*variance / static_cast<double>(n));


    // Free the device memory
    gpuErrchk( hipFree(d_total_value) );
    gpuErrchk( hipFree(d_total_squared_value) );
    gpuErrchk( hipFree(d_function) );
    gpuErrchk( hipFree(d_coefficients) );
    gpuErrchk( hipFree(d_assets_returns) );
    gpuErrchk( hipFree(d_assets_std_devs) );
    gpuErrchk( hipFree(d_assets_last_values) );
    gpuErrchk( hipFree(d_predicted_assets_prices) );
    // hipFree(d_simulated_returns);

    printf("--------->option payoff: %f\n", option_payoff);
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start); // return std::make_pair(69.0, 420.0);
    hipDeviceSynchronize();

    return std::make_pair(option_payoff, static_cast<double>(duration.count()));
}