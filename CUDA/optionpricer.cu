#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

extern "C++"
{
#include "../include/project/hyperrectangle.hpp"
#include "../include/project/asset.hpp"
#include "../include/project/finance_computation.hpp"
#include "../include/project/asset.hpp"
#include "../include/project/finance_montecarlo.hpp"
#include "../include/project/optionparameters.hpp"
#include "../include/project/finance_inputmanager.hpp"
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ void generateGaussianNumbers( float *total_value, float *total_squared_value,
                                        const float *assets_returns, const float *assets_std_devs, long long int n,
                                        float *assets_closing_values, int strike_price, long long int seed, float *predicted_assets_prices )
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float simulated_returns[4]; // TODO 4 is the maximum number of assets
    //size_t num_assets = sizeof(simulated_returns);
    float result = 0.0;
    float rnd_daily_return = 0.0;
    float closing_value;

    if (tid < n)
    {
        hiprandState_t state;
        hiprand_init(seed, tid, 0, &state); // Moved outside the loop

        for( size_t asset_idx = 0; asset_idx < 4; asset_idx++)
        {
            closing_value = assets_closing_values[asset_idx];
            
            float return_value = hiprand_normal(&state);
            rnd_daily_return = assets_returns[asset_idx] + assets_std_devs[asset_idx] * return_value;
            closing_value = closing_value * (1 + rnd_daily_return);

            simulated_returns[asset_idx] = closing_value/assets_closing_values[asset_idx];

            // if (simulated_returns[asset_idx] < assets_returns[asset_idx] - 24 * assets_std_devs[asset_idx] + 1.0){
            //     //printf("Simulated return out of bounds: %f < %f\n", simulated_returns[asset_idx], assets_returns[asset_idx] - 24 * assets_std_devs[asset_idx] + 1.0);
            //     asset_idx--;                
            //     continue;

            // } else if(simulated_returns[asset_idx] > assets_returns[asset_idx] + 24 * assets_std_devs[asset_idx] + 1.0 ){
            //     //printf("Simulated return out of bounds: %f > %f\n", simulated_returns[asset_idx], assets_returns[asset_idx] + 24 * assets_std_devs[asset_idx] + 1.0);
            //     asset_idx--;                
            //     continue;
            // }
            // else {
                result += closing_value;
                atomicAdd(&predicted_assets_prices[asset_idx], closing_value);
                //printf("OK        Simulated return: %f, asset_idx: %d\n", simulated_returns[asset_idx], asset_idx);
            //  }            
            
        }

        if (result > strike_price)
            {
                result = result - strike_price;
            }else result = 0.0;

        atomicAdd(&total_value[tid % 100000], result);
        atomicAdd(&total_squared_value[tid % 100000], result * result);
    }
}




__global__ void printFunction(long long int n, char *function, const double *coefficients, int number_of_coefficients)
{
    printf("Function: %s\n", function);
    printf("n: %d\n", n);
    for (size_t i = 0; i < number_of_coefficients; ++i)
    {
        printf("Coefficient[%d]: %f\n", i, coefficients[i]);
    }
    printf("Number of coefficients: %d\n", number_of_coefficients);
}



extern std::pair<double, double> kernel_wrapper(long long int n, const std::string &function, HyperRectangle &hyperrectangle,
                                                const std::vector<const Asset *> &assetPtrs, double std_dev_from_mean, double *variance,
                                                std::vector<double> coefficients, double strike_price, long long int seed)
{
    auto start = std::chrono::high_resolution_clock::now();
    dim3 threads_per_block = 256;
    dim3 number_of_blocks = (n + threads_per_block.x - 1) / threads_per_block.x;

    uint num_assets = assetPtrs.size();
    // Create and copy function and coefficients to device
    char *d_function;
    size_t function_size = function.size() + 1; // Include the null terminator
    gpuErrchk( hipMalloc((void **)&d_function, function_size * sizeof(char)) );
    gpuErrchk( hipMemcpy(d_function, function.c_str(), function_size * sizeof(char), hipMemcpyHostToDevice) );

    double *d_coefficients;
    gpuErrchk( hipMalloc((void **)&d_coefficients, coefficients.size() * sizeof(double)) );
    gpuErrchk( hipMemcpy(d_coefficients, coefficients.data(), coefficients.size() * sizeof(double), hipMemcpyHostToDevice) );
    

    // Call the CUDA kernel to print the function and coefficients
    // printf("Calling CUDA kernel!\n");
     printFunction<<<1, 1>>>(n, d_function, d_coefficients, coefficients.size());
    // printf("CUDA kernel finished!\n");

    // Save the assets main data
    float *d_assets_returns;
    float *d_assets_std_devs;
    float *d_assets_last_values;
    gpuErrchk( hipMalloc((void **)&d_assets_returns, num_assets * sizeof(float)) );
    gpuErrchk( hipMalloc((void **)&d_assets_std_devs, num_assets * sizeof(float)) );
    gpuErrchk( hipMalloc((void **)&d_assets_last_values, num_assets * sizeof(float)) );

    for (size_t i = 0; i < num_assets; i++)
    {
        float return_mean = static_cast<float>(assetPtrs[i]->getReturnMean());
        gpuErrchk( hipMemcpy(&d_assets_returns[i], &return_mean, sizeof(float), hipMemcpyHostToDevice) );

        float return_std_dev = static_cast<float>(assetPtrs[i]->getReturnStdDev());
        gpuErrchk( hipMemcpy(&d_assets_std_devs[i], &return_std_dev, sizeof(float), hipMemcpyHostToDevice) );

        float last_value = static_cast<float>(assetPtrs[i]->getLastRealValue());
        gpuErrchk( hipMemcpy(&d_assets_last_values[i], &last_value, sizeof(float), hipMemcpyHostToDevice) );
    }

    
    double total_value = 0.0;
    double total_squared_value = 0.0;

    float *d_total_value, *d_total_squared_value;
    gpuErrchk( hipMalloc(&d_total_value, 100000 * sizeof(float)) );
    gpuErrchk( hipMalloc(&d_total_squared_value, 100000 * sizeof(float)) );

    float predicted_assets_prices[num_assets];;
    float *d_predicted_assets_prices;
    gpuErrchk( hipMalloc(&d_predicted_assets_prices, num_assets * sizeof(float)) );

    generateGaussianNumbers<<<number_of_blocks, threads_per_block>>>( d_total_value, d_total_squared_value, d_assets_returns, d_assets_std_devs, n, d_assets_last_values, strike_price, seed, d_predicted_assets_prices);
    hipDeviceSynchronize();

    double domain = 1.0;
    double integration_bounds[num_assets * 2 - 1];
    int j = 0;

        for (size_t i = 0; i < num_assets * 2 - 1; i += 2)
        {
            integration_bounds[i]     = assetPtrs[j]->getReturnMean() - 24 * assetPtrs[j]->getReturnStdDev() + 1.0;
            integration_bounds[i + 1] = assetPtrs[j]->getReturnMean() + 24 * assetPtrs[j]->getReturnStdDev() + 1.0;
            j++;
            domain *= (integration_bounds[i + 1] - integration_bounds[i]);
        }

    float host_total_value[100000];
    float host_total_squared_value[100000];
    gpuErrchk( hipMemcpy(host_total_value, d_total_value, 100000 * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(host_total_squared_value, d_total_squared_value, 100000 * sizeof(float), hipMemcpyDeviceToHost) );

    for (size_t i = 0; i <  100000; i++)
    {
        total_value += static_cast<double>( host_total_value[i] );
        total_squared_value += static_cast<double>( host_total_squared_value[i] );
    }


    float host_assets_prices[num_assets];
    gpuErrchk( hipMemcpy(host_assets_prices, d_predicted_assets_prices, num_assets * sizeof(float), hipMemcpyDeviceToHost) );

    for( size_t i = 0; i < num_assets; ++i )
    {
        predicted_assets_prices[i] = ( host_assets_prices[i]/n );
        std::cout << "The predicted future prices (30 days) of one " << assetPtrs[i]->getName() << " stock is " << predicted_assets_prices[i] << std::endl;
    }


    double integral = total_value / n * domain;

    // calculate the variance
    *variance = total_squared_value/n - (total_value / n) * (total_value/ n);
    *variance = sqrt(*variance / static_cast<double>(n));


    // Free the device memory
    gpuErrchk( hipFree(d_total_value) );
    gpuErrchk( hipFree(d_total_squared_value) );
    gpuErrchk( hipFree(d_function) );
    gpuErrchk( hipFree(d_coefficients) );
    gpuErrchk( hipFree(d_assets_returns) );
    gpuErrchk( hipFree(d_assets_std_devs) );
    gpuErrchk( hipFree(d_assets_last_values) );
    // hipFree(d_simulated_returns);

    printf("--------->Integral: %f\n", integral);
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start); // return std::make_pair(69.0, 420.0);
    hipDeviceSynchronize();

    return std::make_pair(integral, static_cast<double>(duration.count()));
}